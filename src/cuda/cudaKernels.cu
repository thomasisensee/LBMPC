#include "hip/hip_runtime.h"
#include <stdio.h>

#include "cudaKernels.cuh"
#include "cudaErrorHandler.cuh"

#include "core/descriptors/latticeDescriptors.h"
#include "cell.h"
#include "core/constants.h"
#include "core/lb/lbConstants.h"
#include "core/kernelParameters.h"


__device__ unsigned int pos(unsigned int i, unsigned int j, unsigned int width) {
    return j * width + i;
}

template<typename T, unsigned int D, unsigned int Q>
__global__ void initializeDistributionsKernel(T* collision, const CollisionParamsBGK<T>* const params) {

    const unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i > params->Nx - 1 || j > params->Ny - 1) { return; }
    
    unsigned int idx = pos(i, j, params->Nx);
    
    Cell<T,D,Q> cell;
    T R = 1.0;
    T U = 0.0;
    T V = 0.0;

    cell.setEquilibriumDistribution(&collision[Q * idx], R, U, V);
}

template<typename T, unsigned int D, unsigned int Q>
void initializeDistributionsCaller(T* deviceCollision, const CollisionParamsBGK<T>* const params, dim3 gridSize, dim3 blockSize) {
    initializeDistributionsKernel<T,D,Q><<<gridSize, blockSize>>>(deviceCollision, params);
    cudaErrorCheck(hipDeviceSynchronize());
}
template void initializeDistributionsCaller<float,2,9>(float* deviceCollision, const CollisionParamsBGK<float>* const params, dim3 gridSize, dim3 blockSize);
template void initializeDistributionsCaller<float,2,5>(float* deviceCollision, const CollisionParamsBGK<float>* const params, dim3 gridSize, dim3 blockSize);
template void initializeDistributionsCaller<double,2,9>(double* deviceCollision, const CollisionParamsBGK<double>* const params, dim3 gridSize, dim3 blockSize);
template void initializeDistributionsCaller<double,2,5>(double* deviceCollision, const CollisionParamsBGK<double>* const params, dim3 gridSize, dim3 blockSize);

template<typename T, unsigned int D, unsigned int Q>
__global__ void doStreamingKernel(const T *const collision, T *streaming, const CollisionParamsBGK<T>* const params) {

    const unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < 1 || i > params->Nx - 2 || j < 1 || j > params->Ny - 2) { return; }

    unsigned int idx = pos(i, j, params->Nx);
    unsigned int idxNeighbor;
    int cx, cy;

    for (unsigned int l = 0; l < Q; ++l) {
        cx = latticeDescriptors::latticeVelocities<D,Q>(l, 0);
        cy = latticeDescriptors::latticeVelocities<D,Q>(l, 1);

 	    idxNeighbor = pos(static_cast<int>(i) - cx, static_cast<int>(j) - cy, params->Nx);
		streaming[Q * idx + l] = collision[Q * idxNeighbor + l];
	}
}

template<typename T, unsigned int D, unsigned int Q>
void doStreamingCaller(T** deviceCollision, T** deviceStreaming, const CollisionParamsBGK<T>* const params, dim3 gridSize, dim3 blockSize) {
    // Call the cuda kernel
    doStreamingKernel<T,D,Q><<<gridSize, blockSize>>>(*deviceCollision, *deviceStreaming, params);
    cudaErrorCheck(hipDeviceSynchronize());

    // Swap the pointers
    T* swap = *deviceCollision;
    *deviceCollision = *deviceStreaming;
    *deviceStreaming = swap;
}
template void doStreamingCaller<float,2,9>(float** deviceCollision, float** deviceStreaming, const CollisionParamsBGK<float>* const params, dim3 gridSize, dim3 blockSize);
template void doStreamingCaller<float,2,5>(float** deviceCollision, float** deviceStreaming, const CollisionParamsBGK<float>* const params, dim3 gridSize, dim3 blockSize);
template void doStreamingCaller<double,2,9>(double** deviceCollision, double** deviceStreaming, const CollisionParamsBGK<double>* const params, dim3 gridSize, dim3 blockSize);
template void doStreamingCaller<double,2,5>(double** deviceCollision, double** deviceStreaming, const CollisionParamsBGK<double>* const params, dim3 gridSize, dim3 blockSize);

template<typename T, unsigned int D, unsigned int Q>
__global__ void doCollisionBGKKernel(T* collision, const CollisionParamsBGK<T>* const params) {

    const unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < 1 || i > params->Nx - 2 || j < 1 || j > params->Ny - 2) { return; }

    unsigned int idx = pos(i, j, params->Nx);

    Cell<T,D,Q> cell;
    T R = cell.getZerothMoment(&collision[idx * Q]);
    T U = cell.getVelocityX(&collision[idx * Q], R);
    T V = cell.getVelocityY(&collision[idx * Q], R);

    cell.computePostCollisionDistributionBGK(&collision[idx * Q], params, R, U, V);
}


template<typename T, unsigned int D, unsigned int Q>
void doCollisionBGKCaller(T* deviceCollision, const CollisionParamsBGK<T>* const params, dim3 gridSize, dim3 blockSize) {
    doCollisionBGKKernel<T,D,Q><<<gridSize, blockSize>>>(deviceCollision, params);
    cudaErrorCheck(hipDeviceSynchronize());
}
template void doCollisionBGKCaller<float,2,9>(float* deviceCollision, const CollisionParamsBGK<float>* const params, dim3 gridSize, dim3 blockSize);
template void doCollisionBGKCaller<float,2,5>(float* deviceCollision, const CollisionParamsBGK<float>* const params, dim3 gridSize, dim3 blockSize);
template void doCollisionBGKCaller<double,2,9>(double* deviceCollision, const CollisionParamsBGK<double>* const params, dim3 gridSize, dim3 blockSize);
template void doCollisionBGKCaller<double,2,5>(double* deviceCollision, const CollisionParamsBGK<double>* const params, dim3 gridSize, dim3 blockSize);


template<typename T, unsigned int D, unsigned int Q>
__global__ void doCollisionCHMKernel(T* collision, const CollisionParamsCHM<T>* const params) {

    const unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < 1 || i > params->Nx - 2 || j < 1 || j > params->Ny - 2) { return; }

    unsigned int idx = pos(i, j, params->Nx);

    Cell<T,D,Q> cell;
    T R = cell.getZerothMoment(&collision[idx * Q]);
    T U = cell.getVelocityX(&collision[idx * Q], R);
    T V = cell.getVelocityY(&collision[idx * Q], R);

    cell.computePostCollisionDistributionCHM(&collision[idx * Q], params, R, U, V);
}


template<typename T, unsigned int D, unsigned int Q>
void doCollisionCHMCaller(T* deviceCollision, const CollisionParamsCHM<T>* const params, dim3 gridSize, dim3 blockSize) {
    doCollisionCHMKernel<T,D,Q><<<gridSize, blockSize>>>(deviceCollision, params);
    cudaErrorCheck(hipDeviceSynchronize());
}
template void doCollisionCHMCaller<float,2,9>(float* deviceCollision, const CollisionParamsCHM<float>* const params, dim3 gridSize, dim3 blockSize);
template void doCollisionCHMCaller<float,2,5>(float* deviceCollision, const CollisionParamsCHM<float>* const params, dim3 gridSize, dim3 blockSize);
template void doCollisionCHMCaller<double,2,9>(double* deviceCollision, const CollisionParamsCHM<double>* const params, dim3 gridSize, dim3 blockSize);
template void doCollisionCHMCaller<double,2,5>(double* deviceCollision, const CollisionParamsCHM<double>* const params, dim3 gridSize, dim3 blockSize);



template<typename T, unsigned int D, unsigned int Q>
__device__ void applyBC(T* collision, const BoundaryParams<T>* const params, unsigned int i, unsigned int j) {
    unsigned int idx, idxNeighbor, iPop, iPopRev;
    int cix, ciy;
    Cell<T,D,Q> cell;
    T R, dotProduct;

    idx = pos(i, j, params->Nx);

    for (unsigned int l = 0; l < 3; ++l) {
        iPop = latticeDescriptors::boundaryMapping<D,Q>(static_cast<unsigned int>(params->location), l);
        iPopRev = Q - iPop;
        cix = latticeDescriptors::latticeVelocities<D,Q>(iPop, 0);
        ciy = latticeDescriptors::latticeVelocities<D,Q>(iPop, 1);
/*
        if (params->location == BoundaryLocation::EAST) {
            printf("iPop = %d | (cix,ciy) = (%d,%d)\n",iPop,cix,ciy);
        }
*/
        // Check if the neighbor is outside the domain (i.e., a ghost cell)
        if (static_cast<int>(i) + cix < 1 || static_cast<int>(i) + cix > params->Nx - 2 || static_cast<int>(j) + ciy < 1 || static_cast<int>(j) + ciy > params->Ny - 2) { continue; }

        // Compute the index of the neighbor
        idxNeighbor = pos(static_cast<int>(i) + cix, static_cast<int>(j) + ciy, params->Nx);

        // Compute the dot product if WALL_VELOCITY is not null
        if (params->WALL_VELOCITY != nullptr) {
            dotProduct = static_cast<T>(cix) * params->WALL_VELOCITY[0] + static_cast<T>(ciy) * params->WALL_VELOCITY[1];
        } else {
            dotProduct = 0.0;
        }

        // Compute the zeroth moment (i.e., the density) of the neighbor cell
        R = cell.getZerothMoment(&collision[idxNeighbor * Q]);

        // Apply the bounce-back boundary condition
        collision[idx * Q + iPop] = collision[idxNeighbor * Q + iPopRev] + 2.0 * R * dotProduct * latticeDescriptors::latticeWeights<T,D,Q>(iPop) * latticeDescriptors::invCs2<T,D,Q>();
/*
        if (params->location == BoundaryLocation::NORTH) {
            printf("coll = %g, additional term = %g\n",collision[idxNeighbor * Q + iPopRev],2.0 * R * dotProduct * latticeDescriptors::latticeWeights<T,D,Q>(iPop) * latticeDescriptors::invCs2<T,D,Q>());
        }
        */
    }
       
}

template<typename T, unsigned int D, unsigned int Q>
__global__ void applyBounceBackKernel(T* collision, const BoundaryParams<T>* const params) {

    unsigned int i, j;

    switch(params->location) {
    case BoundaryLocation::WEST:
        i = 0;
        j = threadIdx.x + blockIdx.x * blockDim.x;
        if (j > params->Ny - 1) { return; }

        applyBC<T,D,Q>(collision, params, i, j);
        return;
    case BoundaryLocation::EAST:
        i = params->Nx-1;
        j = threadIdx.x + blockIdx.x * blockDim.x;
        if (j > params->Ny - 1) { return; }

        applyBC<T,D,Q>(collision, params, i, j);
        return;     
    case BoundaryLocation::SOUTH:
        i = threadIdx.x + blockIdx.x * blockDim.x;
        j = 0;
        if (i > params->Nx - 1) { return; }

        applyBC<T,D,Q>(collision, params, i, j);
        return;
    case BoundaryLocation::NORTH:
        i = threadIdx.x + blockIdx.x * blockDim.x;
        j = params->Ny-1;
        if (i > params->Nx - 1) { return; }
        applyBC<T,D,Q>(collision, params, i, j);
        return;
    }
}

template<typename T, unsigned int D, unsigned int Q>
void applyBounceBackCaller(T* deviceCollision, const BoundaryParams<T>* const params, dim3 gridSize, dim3 blockSize) {
    applyBounceBackKernel<T,D,Q><<<gridSize, blockSize>>>(deviceCollision, params);
    cudaErrorCheck(hipDeviceSynchronize());
}
template void applyBounceBackCaller<float,2,9>(float* deviceCollision, const BoundaryParams<float>* const params, dim3 gridSize, dim3 blockSize);
template void applyBounceBackCaller<float,2,5>(float* deviceCollision, const BoundaryParams<float>* const params, dim3 gridSize, dim3 blockSize);
template void applyBounceBackCaller<double,2,9>(double* deviceCollision, const BoundaryParams<double>* const params, dim3 gridSize, dim3 blockSize);
template void applyBounceBackCaller<double,2,5>(double* deviceCollision, const BoundaryParams<double>* const params, dim3 gridSize, dim3 blockSize);

template<typename T, unsigned int D, unsigned int Q>
__global__ void computeZerothMomentKernel(T* zerothMoment, const T* const collision, const CollisionParamsBGK<T>* const params) {

    const unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < 1 || i > params->Nx - 2 || j < 1 || j > params->Ny - 2) { return; }

    unsigned int idx        = pos(i, j, params->Nx);
    unsigned int idxMoment  = pos(i - 1, j - 1, params->Nx - 2); // Since the zerothMoment array does not contain the ghost cells

    Cell<T,D,Q> cell;
    T R = cell.getZerothMoment(&collision[idx * Q]);
    zerothMoment[idxMoment] = R;
}

template<typename T, unsigned int D, unsigned int Q>
void computeZerothMomentCaller(T* deviceZerothMoment, const T* const deviceCollision, const CollisionParamsBGK<T>* const params, dim3 gridSize, dim3 blockSize) {
    computeZerothMomentKernel<T,D,Q><<<gridSize, blockSize>>>(deviceZerothMoment, deviceCollision, params);
    cudaErrorCheck(hipDeviceSynchronize());
}
template void computeZerothMomentCaller<float,2,9>(float* deviceZerothMoment, const float* const deviceCollision, const CollisionParamsBGK<float>* const params, dim3 gridSize, dim3 blockSize);
template void computeZerothMomentCaller<float,2,5>(float* deviceZerothMoment, const float* const deviceCollision, const CollisionParamsBGK<float>* const params, dim3 gridSize, dim3 blockSize);
template void computeZerothMomentCaller<double,2,9>(double* deviceZerothMoment, const double* const deviceCollision, const CollisionParamsBGK<double>* const params, dim3 gridSize, dim3 blockSize);
template void computeZerothMomentCaller<double,2,5>(double* deviceZerothMoment, const double* const deviceCollision, const CollisionParamsBGK<double>* const params, dim3 gridSize, dim3 blockSize);

template<typename T, unsigned int D, unsigned int Q>
__global__ void computeFirstMomentKernel(T* firstMoment, const T* const collision, const CollisionParamsBGK<T>* const params) {

    const unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < 1 || i > params->Nx - 2 || j < 1 || j > params->Ny - 2) { return; }

    unsigned int idx        = pos(i, j, params->Nx);
    unsigned int idxMoment  = pos(i - 1, j - 1, params->Nx - 2); // Since the zerothMoment array does not contain the ghost cells

    Cell<T,D,Q> cell;
    T U = cell.getFirstMomentX(&collision[idx * Q]);
    T V = cell.getFirstMomentY(&collision[idx * Q]);

    firstMoment[idxMoment * D]      = U;
    firstMoment[idxMoment * D + 1]  = V;
}

template<typename T, unsigned int D, unsigned int Q>
void computeFirstMomentCaller(T* deviceFirstMoment, const T* const deviceCollision, const CollisionParamsBGK<T>* const params, dim3 gridSize, dim3 blockSize) {
    computeFirstMomentKernel<T,D,Q><<<gridSize, blockSize>>>(deviceFirstMoment, deviceCollision, params);
    cudaErrorCheck(hipDeviceSynchronize());
}
template void computeFirstMomentCaller<float,2,9>(float* deviceFirstMoment, const float* const deviceCollision, const CollisionParamsBGK<float>* const params, dim3 gridSize, dim3 blockSize);
template void computeFirstMomentCaller<float,2,5>(float* deviceFirstMoment, const float* const deviceCollision, const CollisionParamsBGK<float>* const params, dim3 gridSize, dim3 blockSize);
template void computeFirstMomentCaller<double,2,9>(double* deviceFirstMoment, const double* const deviceCollision, const CollisionParamsBGK<double>* const params, dim3 gridSize, dim3 blockSize);
template void computeFirstMomentCaller<double,2,5>(double* deviceFirstMoment, const double* const deviceCollision, const CollisionParamsBGK<double>* const params, dim3 gridSize, dim3 blockSize);