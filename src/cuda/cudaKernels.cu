#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "core/constants.h"
#include "core/kernelParameters.h"
#include "core/cell.h"
#include "cudaKernels.cuh"
#include "cudaErrorHandler.cuh"

__device__ unsigned int pos(unsigned int i, unsigned int j, unsigned int width) {
    return j * width + i;
}

template<typename T>
__global__ void initializeDistributionsKernel(T* collision, const LBParams<T>* const params) {

    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i>params->Nx || j > params->Ny) { return; }
    
    unsigned int idx = pos(i, j, params->Nx);
    
    Cell<T> cell;
    T R = 1;
    T U = 0;
    T V = 0;
    
    cell.setEquilibriumDistribution(&collision[params->Q*idx], params, R, U, V);
}

template<typename T>
void initializeDistributionsCaller(T* deviceCollision, const LBParams<T>* const params, dim3 gridSize, dim3 blockSize) {
    initializeDistributionsKernel<<<gridSize, blockSize>>>(deviceCollision, params);
    cudaErrorCheck(hipDeviceSynchronize());
}
template void initializeDistributionsCaller<float>(float* deviceCollision, const LBParams<float>* const params, dim3 gridSize, dim3 blockSize);
template void initializeDistributionsCaller<double>(double* deviceCollision, const LBParams<double>* const params, dim3 gridSize, dim3 blockSize);

template<typename T>
__global__ void doStreamingKernel(const T *const collision, T *streaming, const LBParams<T>* const params) {

    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i<1 || i>params->Nx-1 || j<1 || j > params->Ny-1) { return; }

    unsigned int idx = pos(i, j, params->Nx);
    unsigned int idxNeighbor;

    for (int l=0; l<params->Q; l++) {
	    idxNeighbor = pos(i-params->LATTICE_VELOCITIES[l*2],j-params->LATTICE_VELOCITIES[l*2+1],params->Nx);
		streaming[params->Q*idx+l]=collision[params->Q*idxNeighbor];
	}
}

template<typename T>
void doStreamingCaller(T* deviceCollision, T* deviceStreaming, T* swap, const LBParams<T>* const params, dim3 gridSize, dim3 blockSize) {
    doStreamingKernel<<<gridSize, blockSize>>>(deviceCollision, deviceStreaming, params);
    cudaErrorCheck(hipDeviceSynchronize());
    swap = deviceCollision; deviceCollision = deviceStreaming; deviceStreaming = swap;
}
template void doStreamingCaller<float>(float* deviceCollision, float* deviceStreaming, float* swap, const LBParams<float>* const params, dim3 gridSize, dim3 blockSize);
template void doStreamingCaller<double>(double* deviceCollision, double* deviceStreaming, double* swap, const LBParams<double>* const params, dim3 gridSize, dim3 blockSize);


template<typename T>
__global__ void doCollisionCHMKernel(T* collision, const CollisionParamsCHM<T>* const params) {

    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i<1 || i>params->Nx-1 || j<1 || j > params->Ny-1) { return; }

    unsigned int idx = pos(i, j, params->Nx);

    Cell<T> cell;
    T R = cell.getZeroMoment(&collision[params->Q*idx], params);
    T U = cell.getVelocityX(&collision[params->Q*idx], params, R);
    T V = cell.getVelocityY(&collision[params->Q*idx], params, R);   

    cell.computePostCollisionDistributionCHM(&collision[params->Q*idx], params, R, U, V);
}

template<typename T>
void doCollisionCHMCaller(T* deviceCollision, const CollisionParamsCHM<T>* const params, dim3 gridSize, dim3 blockSize) {
    doCollisionCHMKernel<<<gridSize, blockSize>>>(deviceCollision, params);
    cudaErrorCheck(hipDeviceSynchronize());
}
template void doCollisionCHMCaller<float>(float* deviceCollision, const CollisionParamsCHM<float>* const params, dim3 gridSize, dim3 blockSize);
template void doCollisionCHMCaller<double>(double* deviceCollision, const CollisionParamsCHM<double>* const params, dim3 gridSize, dim3 blockSize);

template<typename T>
__device__ void applyBC(T* collision, const BoundaryParams<T>* const params, unsigned int i, unsigned int j) {
    unsigned int idx, idxNeighbor, pop, popRev;
    int cx, cy;
    Cell<T> cell;
    T R, dotProduct;

    idx = pos(i, j, params->Nx);
        
    for (int l = 0; l<3; ++l) {
        pop = params->POPULATION[l];
        popRev = params->OPPOSITE_POPULATION[pop];
        cx = params->LATTICE_VELOCITIES[pop*params->D];
        cy = params->LATTICE_VELOCITIES[pop*params->D+1];
        if ((int) i+cx < 0 || i+cx > params->Nx || (int) j+cy < 0 || j+cy > params->Ny) { continue; }
        idxNeighbor = pos(i+cx, j+cy, params->Nx);
        if (params->WALL_VELOCITY != nullptr) {
            dotProduct = cx*params->WALL_VELOCITY[0] + cy*params->WALL_VELOCITY[1];
        } else {
            dotProduct = 0.0;
        }
        R = cell.getZeroMoment(&collision[params->Q*idxNeighbor], params);
        printf("(%d,%d) -> (%d,%d) | pop = %d | popRev = %d | CX = %d, CY = %d |idxNeibor = %d | R = %g | dotProduct = %g\n",i,j,i+cx,j+cy,pop,popRev,params->LATTICE_VELOCITIES[pop*params->D],params->LATTICE_VELOCITIES[pop*params->D+1],idxNeighbor,R,dotProduct);
        
        collision[params->Q*idx + pop] = collision[params->Q*idxNeighbor + popRev] + 2.0*params->LATTICE_WEIGHTS[pop] * R * C_S_POW2_INV * dotProduct;
    }
       
}

template<typename T>
__global__ void applyBounceBackKernel(T* collision, const BoundaryParams<T>* const params) {

    unsigned int i, j;

    switch(params->location) {
    case BoundaryLocation::WEST:
        i = 0;
        j = threadIdx.x + blockIdx.x * blockDim.x;
        if (j > params->Ny-1) { return; }

        applyBC(collision, params, i, j);
        return;
        
        
        /*
    case BoundaryLocation::EAST:
        i = params->Nx+1;
        j = threadIdx.x + blockIdx.x * blockDim.x;
        if (j > params->Ny-1) { return; }
        
        idx = pos(i, j, params->Nx);
        return;
    case BoundaryLocation::SOUTH:
        i = threadIdx.x + blockIdx.x * blockDim.x;
        j = 0;
        if (i > params->Ny-1) { return; }
        
        idx = pos(i, j, params->Nx);
        return;
    case BoundaryLocation::NORTH:
        i = threadIdx.x + blockIdx.x * blockDim.x;
        j = params->Ny-1;
        if (i > params->Ny-1) { return; }

        idx = pos(i, j, params->Nx);
        return;
       */ 
    }

}

template<typename T>
void applyBounceBackCaller(T* deviceCollision, const BoundaryParams<T>* const params, dim3 gridSize, dim3 blockSize) {
    applyBounceBackKernel<<<gridSize, blockSize>>>(deviceCollision, params);
    cudaErrorCheck(hipDeviceSynchronize());
}
template void applyBounceBackCaller<float>(float* deviceCollision, const BoundaryParams<float>* const params, dim3 gridSize, dim3 blockSize);
template void applyBounceBackCaller<double>(double* deviceCollision, const BoundaryParams<double>* const params, dim3 gridSize, dim3 blockSize);

